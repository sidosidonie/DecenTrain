#include <hip/hip_runtime.h>
#include <iostream>

#define CHECK_CUDA(call)                                                        \
    do {                                                                        \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error in " << __FILE__ << "@" << __LINE__       \
                      << ": " << hipGetErrorString(err) << std::endl;         \
            exit(EXIT_FAILURE);                                                \
        }                                                                       \
    } while (0)

int main() {
    const int dataSize = 100 * 1024 * 1024;  // 100 MB
    const size_t size = dataSize * sizeof(char);

    char *h_src, *h_dst;
    char *d_src, *d_dst;

    // Allocate host memory
    //h_src = (char *)malloc(size);
    //h_dst = (char *)malloc(size);
    hipHostAlloc((void**)&h_src, size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_dst, size, hipHostMallocDefault);

    // Allocate device memory
    CHECK_CUDA(hipMalloc((void **)&d_src, size));
    CHECK_CUDA(hipMalloc((void **)&d_dst, size));

    // Fill source data
    memset(h_src, 1, size);

    hipEvent_t start, stop;
    float elapsedTime;

    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Host to Device
    CHECK_CUDA(hipEventRecord(start, 0));
    CHECK_CUDA(hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "Host to Device bandwidth: " << (size / (elapsedTime * 1e6)) << " GB/s\n";

    // Device to Host
    CHECK_CUDA(hipEventRecord(start, 0));
    CHECK_CUDA(hipMemcpy(h_dst, d_src, size, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "Device to Host bandwidth: " << (size / (elapsedTime * 1e6)) << " GB/s\n";

    // Device to Device
    CHECK_CUDA(hipEventRecord(start, 0));
    CHECK_CUDA(hipMemcpy(d_dst, d_src, size, hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&elapsedTime, start, stop));
    std::cout << "Device to Device bandwidth: " << (size / (elapsedTime * 1e6)) << " GB/s\n";

    // Cleanup
    hipHostFree(h_src);
    hipHostFree(h_dst);
    hipFree(d_src);
    hipFree(d_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
